// ---------------------------------------------------------
// Copyright (c) 2016, Andy Zeng
// 
// This file is part of the APC Vision Toolbox and is available 
// under the terms of the Simplified BSD License provided in 
// LICENSE. Please retain this notice and LICENSE if you use 
// this file (or any portion of it) in your project.
// ---------------------------------------------------------

#include "depth_utils.h"
#include "ros/ros.h"
#include "marvin_convnet/DetectObjects.h"
#include "marvin_convnet/object_mask.h"
//#include "realsense_camera/StreamSensor.h"
#include <opencv2/opencv.hpp>

// Marvin
#define DATATYPE 0
#include "marvin.hpp"

#include <sensor_msgs/image_encodings.h>
#include <image_transport/image_transport.h>
#include <cv_bridge/cv_bridge.h>




//std::string shelf_net_arch_filename = "/home/joinet/text-pick-n-place-baseline/ros-packages/catkin_ws/src/marvin_convnet/models/competition/net.json";
std::string tote_net_arch_filename = "/home/joinet/text-pick-n-place-baseline/ros-packages/catkin_ws/src/marvin_convnet/models/competition/net.json";
//std::string shelf_net_weights_filename = "/home/joinet/text-pick-n-place-baseline/ros-packages/catkin_ws/src/marvin_convnet/models/competition/weights_shelf.marvin";
std::string tote_net_weights_filename = "/home/joinet/text-pick-n-place-baseline/ros-packages/catkin_ws/src/marvin_convnet/models/competition/weights_tote.marvin";


// Service modes and names
std::string service_name;

// Directory to read/write all RGB-D files and response maps
std::string read_directory;

// Global buffers for sensor data retrieval
int frame_width = 640;
int frame_height = 480;
uint8_t * color_buffer = new uint8_t[frame_width * frame_height * 3];
uint8_t * HHA_buffer = new uint8_t[frame_width * frame_height * 3];

// Load Marvin FCN network architectures
//marvin::Net shelf_net(shelf_net_arch_filename);
marvin::Net tote_net(tote_net_arch_filename);

// Marvin responses
StorageT* color_data_CPU = NULL;
StorageT* HHA_data_CPU = NULL;
StorageT* prob_CPU_StorageT = NULL;
ComputeT* prob_CPU_ComputeT = NULL;

ros::ServiceClient client_sensor;

const int num_apc_objects = 39;

std::string shelf_bin_ids = "ABCDEFGHIJKL";


// For realtime prediction
class ImageConverter
{
  ros::NodeHandle nh_;
  ros::NodeHandle nh_p;
  ros::Publisher pub = nh_p.advertise<sensor_msgs::Image>("/mask_prediction", 1);

  ros::NodeHandle nh_ps;
  ros::Publisher pub_s = nh_ps.advertise<marvin_convnet::object_mask>("/mask_prediction_with_class", 1);

  marvin_convnet::object_mask object_mask;


  image_transport::ImageTransport it_;
  image_transport::Subscriber image_sub_;
  image_transport::Publisher image_pub_;

public:
  ImageConverter()
    : it_(nh_)
  {
    // Subscrive to input video feed and publish output video feed
    image_sub_ = it_.subscribe("/camera/rgb/image_raw", 1, &ImageConverter::imageCb, this);
    

    //cv::namedWindow(OPENCV_WINDOW);
  }

  ~ImageConverter()
  {
    //cv::destroyWindow(OPENCV_WINDOW);
  }

  void imageCb(const sensor_msgs::ImageConstPtr& msg)
  {

    cv_bridge::CvImagePtr cv_ptr;
    cv::Mat color_frame;
    
    try
    {
      //cv_ptr = cv_bridge::toCvCopy(msg, sensor_msgs::image_encodings::BGR8);
      color_frame = cv_bridge::toCvShare(msg, "bgr8")->image;
    }
    catch (cv_bridge::Exception& e)
    {
      ROS_ERROR("cv_bridge exception: %s", e.what());
      return;
    }

    // Draw an example circle on the video stream
    //if (cv_ptr->image.rows > 60 && cv_ptr->image.cols > 60)
      //cv::circle(cv_ptr->image, cv::Point(50, 50), 10, CV_RGB(255,0,0));

      color_buffer = color_frame.data;
       
      for (int r = 0; r < frame_height; ++r)
    	  for (int c = 0; c < frame_width; ++c) {
      color_data_CPU[0 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(color_buffer[0 + 3 * (c + frame_width * r)]) - ComputeT(102.9801f)); // B
      color_data_CPU[1 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(color_buffer[1 + 3 * (c + frame_width * r)]) - ComputeT(115.9465f)); // G
      color_data_CPU[2 * frame_height * frame_width + r * frame_width + c] = CPUCompute2StorageT(ComputeT(color_buffer[2 + 3 * (c + frame_width * r)]) - ComputeT(122.7717f)); // R
      } 



	  // Run forward pass through marvin FCN
	  ROS_INFO("Forward Marvin to get segmentation results.");
	  marvin::Response * rDataRGB;
	  marvin::Response * rDataHHA;
	  marvin::Response * rProb;
          int bin_id = -1; //force to use tote model
	  if (bin_id == -1) {
	    rDataRGB = tote_net.getResponse("data_RGB");
	    rProb = tote_net.getResponse("prob");
	  } else {
	    //rDataRGB = shelf_net.getResponse("data_RGB");
	    //rProb = shelf_net.getResponse("prob");
	  }
	  hipMemcpy(rDataRGB->dataGPU, color_data_CPU, rDataRGB->numBytes(), hipMemcpyHostToDevice);
	  if (bin_id == -1)
	    tote_net.forward();
	  //else
	    //shelf_net.forward();
	  hipMemcpy(prob_CPU_StorageT, rProb->dataGPU, rProb->numBytes(), hipMemcpyDeviceToHost);
	  for (int i = 0; i < frame_height * frame_width * (num_apc_objects + 1); ++i)
	    prob_CPU_ComputeT[i] = CPUStorage2ComputeT(prob_CPU_StorageT[i]);

  // Get full object list
  std::vector<std::string> all_object_names = {"background", "barkely_hide_bones", "cherokee_easy_tee_shirt", "clorox_utility_brush", "cloud_b_plush_bear", "command_hooks", "cool_shot_glue_sticks", "crayola_24_ct", "creativity_chenille_stems", "dasani_water_bottle",
                                               "dove_beauty_bar", "dr_browns_bottle_brush", "easter_turtle_sippy_cup", "elmers_washable_no_run_school_glue", "expo_dry_erase_board_eraser", "fiskars_scissors_red", "fitness_gear_3lb_dumbbell", "folgers_classic_roast_coffee", "hanes_tube_socks", "i_am_a_bunny_book",
                                               "jane_eyre_dvd", "kleenex_paper_towels", "kleenex_tissue_box", "kyjen_squeakin_eggs_plush_puppies", "laugh_out_loud_joke_book", "oral_b_toothbrush_green", "oral_b_toothbrush_red", "peva_shower_curtain_liner", "platinum_pets_dog_bowl", "rawlings_baseball",
                                               "rolodex_jumbo_pencil_cup", "safety_first_outlet_plugs", "scotch_bubble_mailer", "scotch_duct_tape", "soft_white_lightbulb", "staples_index_cards", "ticonderoga_12_pencils", "up_glucose_bottle", "womens_knit_gloves", "woods_extension_cord"};

 std::vector<std::string> selected_object_names = all_object_names;  // do all the objects segmetation
  

  // Loop through each object in selected list
  for (int selected_idx = 0; selected_idx < selected_object_names.size(); selected_idx++) {
    std::string curr_object_name = selected_object_names[selected_idx];
    int curr_object_idx = std::distance(all_object_names.begin(), find(all_object_names.begin(), all_object_names.end(), curr_object_name));
    std::vector<ComputeT> predMap_object(prob_CPU_ComputeT + curr_object_idx * frame_height * frame_width, prob_CPU_ComputeT + (curr_object_idx + 1) * frame_height * frame_width);

    // Create a folder to save results
    //std::ifstream file(read_directory + "/masks");
    //if (file.fail())
      //system(std::string("mkdir -p " + read_directory + "/masks").c_str());

    // Write segmentation response maps to 16-bit grayscale png image
    //std::string result_filename = read_directory + "/masks/frame-" + frame_prefix.str() + "." + all_object_names[curr_object_idx] + ".mask.png";
    cv::Mat result_mat(frame_height, frame_width, CV_8UC1);
    for (size_t y = 0; y < frame_height; y++)
      for (size_t x = 0; x < frame_width; x++) {
        //unsigned short depth_short = (unsigned short)(predMap_object[y * frame_width + x] * 65535);
        //result_mat.at<unsigned short>(y, x) = depth_short;
        unsigned char depth_char = (unsigned char)(predMap_object[y * frame_width + x] * 255);
        result_mat.at<unsigned char>(y, x) = depth_char;

      }

    cv_bridge::CvImage cv_image;
    cv::Mat result_mat_final(480, 640, CV_16UC1);
    result_mat_final = result_mat;

    cv_image.image = result_mat_final;
    cv_image.encoding = "mono8";
    sensor_msgs::Image ros_image;
    cv_image.toImageMsg(ros_image);
    
    
    std::stringstream ss;
    ss << "/mask_prediction/" << selected_idx; //asign specific topic to prediction
    std::string s_object = ss.str();
    
    object_mask.probmask = ros_image;
    object_mask.object = selected_object_names[selected_idx];
    pub_s.publish(object_mask);


    pub.publish(ros_image);
  }


  }


};

















int main(int argc, char **argv) {

  // Setup ROS
  ros::init(argc, argv, "marvin_convnet", ros::init_options::AnonymousName);
  //ros::NodeHandle n;
  //ros::NodeHandle priv_nh("~");

  // Get service parameters
  //priv_nh.param("service_name", service_name, std::string("marvin_convnet"));
  //priv_nh.param("read_directory", read_directory, std::string(""));

  // Assert parameters and create folder to save segmentation masks
  //assert(!read_directory.empty());
  //system(std::string("mkdir -p " + read_directory).c_str());

  // Start service
  //ros::ServiceServer service_detect = n.advertiseService(service_name, srv_detect);

  // Connect to Realsense camera
  //ROS_INFO("Reading data from directory: %s", read_directory.c_str());

  // Setup Marvin
  ROS_INFO("Loading Marvin.");
  //shelf_net.Malloc(marvin::Testing);
  tote_net.Malloc(marvin::Testing);
  //shelf_net.loadWeights(shelf_net_weights_filename);
  tote_net.loadWeights(tote_net_weights_filename);
  color_data_CPU = new StorageT[frame_width * frame_height * 3];
  HHA_data_CPU = new StorageT[frame_width * frame_height * 3];
  prob_CPU_StorageT = new StorageT[frame_width * frame_height * (num_apc_objects + 1)];
  prob_CPU_ComputeT = new ComputeT[frame_height * frame_width * (num_apc_objects + 1)];


  ImageConverter ic;
  ROS_INFO("Ready.");
  ros::spin();

  return 0;
}


